#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <unistd.h>
#include <signal.h>

#include "utils.h"

#define BLOCK_WIDTH 32
#define GRID_SIZE 50

bool continueNextGeneration = true;

/**
 * @brief Determine whether coordinates are within the bounds of shared memory.
 * 
 * @param x x coordinate
 * @param y y coordinate
 * @return boolean
 */
__device__ bool useShared(int x, int y) {
  if (x < 0 || y < 0 || x >= blockDim.x || y >= blockDim.y) {
    return false;
  }
  return true;
}

__device__ int getIdx_global(int x, int y) {
  if (x < 0 || y < 0 || x >= GRID_SIZE || y >= GRID_SIZE) {
    return -1;
  }
  return y * GRID_SIZE + x;
}

__device__ int getIdx_shared(int x, int y) {
  return y * blockDim.x + x;
}

__device__ bool assessNeighborCount(int counter, bool currentVal) {
  if (counter < 2) {
    return false;
  }
  if (currentVal) {
    if (counter <= 3) {
      // Survives
      return true;
    }
    // Dies
    return false;
  }
  
  // Cell is currently dead - will it become alive?
  if (counter == 3) {
    // Yes
    return true;
  }

  // No
  return false;
}

__global__ void computeNextGeneration(const bool* const inputGrid, bool* const outputGrid) {
  int2 pos2d = make_int2(blockDim.x * blockIdx.x + threadIdx.x, blockDim.y * blockIdx.y + threadIdx.y);
  int idx = pos2d.y * GRID_SIZE + pos2d.x;

  if (pos2d.x >= GRID_SIZE || pos2d.y > GRID_SIZE) {
    return;
  }

  // Obtain mapped shared memory location
  int2 s_pos2d = make_int2(threadIdx.x, threadIdx.y);
  int s_idx = s_pos2d.y * blockDim.x + s_pos2d.x;

  __shared__ bool s_inputGrid[BLOCK_WIDTH * BLOCK_WIDTH];
  s_inputGrid[s_idx] = inputGrid[idx];
  __syncthreads();

  int counter = 0;
  int cellValue;

  // Top left
  int topLeftIdx;
  if (useShared(s_pos2d.x - 1, s_pos2d.y - 1)) {
    topLeftIdx = getIdx_shared(s_pos2d.x - 1, s_pos2d.y - 1);
    counter += s_inputGrid[topLeftIdx] ? 1 : 0;
  } else {
    topLeftIdx = getIdx_global(pos2d.x - 1, pos2d.y - 1);
    if (topLeftIdx >= 0) {
      counter += inputGrid[topLeftIdx] ? 1 : 0;
    }
  }

  // Top
  int topIdx;
  if (useShared(s_pos2d.x, s_pos2d.y - 1)) {
    topIdx = getIdx_shared(s_pos2d.x, s_pos2d.y - 1);
    counter += s_inputGrid[topIdx] ? 1 : 0;
  } else {
    topIdx = getIdx_global(pos2d.x, pos2d.y - 1);
    if (topIdx >= 0) {
      counter += inputGrid[topIdx] ? 1 : 0;
    }
  }

  // Top right
  int topRightIdx;
  if (useShared(s_pos2d.x + 1, s_pos2d.y - 1)) {
    topRightIdx = getIdx_shared(s_pos2d.x + 1, s_pos2d.y - 1);
    counter += s_inputGrid[topRightIdx] ? 1 : 0;
  } else {
    topRightIdx = getIdx_global(pos2d.x + 1, pos2d.y - 1);
    if (topRightIdx >= 0) {
      counter += inputGrid[topRightIdx] ? 1 : 0;
    }
  }

  // Left
  int leftIdx;
  if (useShared(s_pos2d.x - 1, s_pos2d.y)) {
    leftIdx = getIdx_shared(s_pos2d.x - 1, s_pos2d.y);
    counter += s_inputGrid[leftIdx] ? 1 : 0;
  } else {
    leftIdx = getIdx_global(pos2d.x - 1, pos2d.y);
    if (leftIdx >= 0) {
      counter += inputGrid[leftIdx] ? 1 : 0;
    }
  }

  // Right
  int rightIdx;
  if (useShared(s_pos2d.x + 1, s_pos2d.y)) {
    rightIdx = getIdx_shared(s_pos2d.x + 1, s_pos2d.y);
    counter += s_inputGrid[rightIdx] ? 1 : 0;
  } else {
    rightIdx = getIdx_global(pos2d.x + 1, pos2d.y);
    if (rightIdx >= 0) {
      counter += inputGrid[rightIdx] ? 1 : 0;
    }
  }

  // Bottom left
  int bottomLeftIdx;
  if (useShared(s_pos2d.x - 1, s_pos2d.y + 1)) {
    bottomLeftIdx = getIdx_shared(s_pos2d.x - 1, s_pos2d.y + 1);
    counter += s_inputGrid[bottomLeftIdx] ? 1 : 0;
  } else {
    bottomLeftIdx = getIdx_global(pos2d.x - 1, pos2d.y + 1);
    if (bottomLeftIdx >= 0) {
      counter += inputGrid[bottomLeftIdx] ? 1 : 0;
    }
  }
  
  // Bottom
  int bottomIdx;
  if (useShared(s_pos2d.x, s_pos2d.y + 1)) {
    bottomIdx = getIdx_shared(s_pos2d.x, s_pos2d.y + 1);
    counter += s_inputGrid[bottomIdx] ? 1 : 0;
  } else {
    bottomIdx = getIdx_global(pos2d.x, pos2d.y + 1);
    if (bottomIdx >= 0) {
      counter += inputGrid[bottomIdx] ? 1 : 0;
    }
  }

  // Bottom right
  int bottomRightIdx;
  if (useShared(s_pos2d.x + 1, s_pos2d.y + 1)) {
    bottomRightIdx = getIdx_shared(s_pos2d.x + 1, s_pos2d.y + 1);
    counter += s_inputGrid[bottomRightIdx] ? 1 : 0;
  } else {
    bottomRightIdx = getIdx_global(pos2d.x + 1, pos2d.y + 1);
    if (bottomRightIdx >= 0) {
      counter += inputGrid[bottomRightIdx] ? 1 : 0;
    }
  }

  // Assess counter value
  outputGrid[idx] = assessNeighborCount(counter, s_inputGrid[s_idx]);
}

__global__ void placeCells(bool* const grid, hiprandState* state) {
  const int2 twoDimCoords = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
                                      blockIdx.y * blockDim.y + threadIdx.y);
  const int idx = twoDimCoords.y * GRID_SIZE + twoDimCoords.x;

  if (twoDimCoords.x >= GRID_SIZE || twoDimCoords.y >= GRID_SIZE) {
    return;
  }
  int randomVal = (int)(hiprand_uniform(&state[idx]) * 100.0f);
  if (randomVal > 75) {
    grid[idx] = 1;
  } else {
    grid[idx] = 0;
  }
}

__global__ void initCurand(hiprandState* state, unsigned long seed) {
  int2 pos2d = make_int2(threadIdx.x + blockDim.x * blockIdx.x, threadIdx.y + blockDim.y * blockIdx.y);
  int idx = pos2d.y * GRID_SIZE + pos2d.x;

  hiprand_init(seed, idx, 0, &state[idx]);
}

void printGrid(bool* grid) {
  printf("---------------\n");
  for (int i = 0; i < GRID_SIZE; ++i) {
    for (int j = 0; j < GRID_SIZE; ++j) {
      int idx = i * GRID_SIZE + j;
      char*  val;
      if (grid[idx]) {
        val = "#";
      } else {
        val = "-";
      }
      std::cout << val << " ";
    }
    std::cout << "\n";
  }
}

void handleSignal(int sigNum) {
  continueNextGeneration = false;
}

int main(int argc, char** argv) {
  const dim3 blockSize(BLOCK_WIDTH, BLOCK_WIDTH);
  // const dim3 gridSize(1,1,1);
  const dim3 gridSize(ceil(1.0f*GRID_SIZE / blockSize.x), ceil(1.0f*GRID_SIZE / blockSize.y));
  gpuErrchk(hipFree(0));

  size_t allocSize = sizeof(bool) * GRID_SIZE * GRID_SIZE;
  bool* d_gridInput;
  bool* d_gridOutput;
  gpuErrchk(hipMalloc(&d_gridInput, allocSize));
  gpuErrchk(hipMalloc(&d_gridOutput, allocSize));

  unsigned long seed = time(NULL);

  hiprandState* state;
  gpuErrchk(hipMalloc((void **)&state, sizeof(hiprandState) * GRID_SIZE * GRID_SIZE));

  initCurand<<<gridSize, blockSize>>>(state, seed);

  hipDeviceSynchronize();
  gpuErrchk(hipGetLastError());

  placeCells<<<gridSize, blockSize>>>(d_gridInput, state);
  hipDeviceSynchronize();
  gpuErrchk(hipGetLastError());

  bool* initialGrid = (bool*)malloc(allocSize);
  gpuErrchk(hipMemcpy(initialGrid, d_gridInput, allocSize, hipMemcpyDeviceToHost));
  printGrid(initialGrid);
  free(initialGrid);

  signal(SIGINT, handleSignal);

  while (continueNextGeneration) {
    bool* output = (bool*)malloc(allocSize);
    computeNextGeneration<<<gridSize, blockSize>>>(d_gridInput, d_gridOutput);
    hipDeviceSynchronize();

    gpuErrchk(hipGetLastError());
    gpuErrchk(hipMemcpy(output, d_gridOutput, allocSize, hipMemcpyDeviceToHost));
    printGrid(output);

    gpuErrchk(hipMemcpy(d_gridInput, d_gridOutput, allocSize, hipMemcpyDeviceToDevice));
    gpuErrchk(hipMemset(d_gridOutput, 0, allocSize));

    free(output);
    // sleep(1);
    struct timespec tim, tim2;
    tim.tv_sec = 0;
    tim.tv_nsec = 100000000L;
    nanosleep(&tim, &tim2);
  }

  gpuErrchk(hipFree(state));
  gpuErrchk(hipFree(d_gridInput));
  gpuErrchk(hipFree(d_gridOutput));

  printf("\nExiting.\n");
}